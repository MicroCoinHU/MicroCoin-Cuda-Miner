#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <cstdint>
#include ""
#include <chrono>

#include <stdio.h>
#include <stdlib.h>
#include <Windows.h>
#include <ctime>

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif

hipStream_t cudastream;
static uint32_t hextarpow = 0xFFFFFFFF;
static char tarpow[11];
uint32_t *blockHeadermobj = nullptr;
uint32_t *midStatemobj = nullptr;
uint32_t *nonceOutmobj = nullptr;

__device__ __forceinline__ uint32_t ror(const uint32_t a, const unsigned int n)
{
#if __CUDA_ARCH__ >= 350
	return __funnelshift_r((a), (a), (n));
#else
	return (a >> n) | (a << (32 - n));
#endif
}


#define shr (a) >> (n);

#define ROTRIGHT(a,b) ((a >> b) | (a << (32 - b)))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))
#define SIG0c(x) (ror(x,7) ^ ror(x,18) ^ ((x) >> 3))
#define SIG1c(x) (ror(x,17) ^ ror(x,19) ^ ((x) >> 10))

#define blocksize 2048 
#define npt 9

static const uint32_t k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
	0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
	0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
	0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
	0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
	0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
	0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
	0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
	0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

__global__ void __launch_bounds__(blocksize, 8) nonceGrindc(uint32_t *const __restrict__ headerIn, uint32_t *const __restrict__ midstateIn, uint32_t *const __restrict__ nonceOut, uint32_t const hextarpowpo)
{
	static const uint32_t k[64] = {
		0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
		0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
		0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
		0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
		0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
		0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
		0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
		0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
		0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
		0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
		0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
		0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
		0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
		0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
		0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
		0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
	};

	// int i = (blockIdx.x * blockDim.x * blockDim.y) + threadIdx.x;
	//i *= npt;

	uint32_t buffer[16];
	uint32_t midstate[8];

	// const uint32_t id = (blockDim.x * blockIdx.x * blockIdx.x * blockIdx.x * headerIn[16] + threadIdx.x)*npt;
	const uint32_t id = (headerIn[16] << 20) | (blockDim.x * blockIdx.x + threadIdx.x)*npt;

	midstate[0] = midstateIn[0];
	midstate[1] = midstateIn[1];
	midstate[2] = midstateIn[2];
	midstate[3] = midstateIn[3];
	midstate[4] = midstateIn[4];
	midstate[5] = midstateIn[5];
	midstate[6] = midstateIn[6];
	midstate[7] = midstateIn[7];

	int j = 0;

#pragma unroll
	for (j = 0; j < 16; j++)
	{
		buffer[j] = headerIn[j];
	}

	uint32_t block[64];

	uint32_t temp1;
	uint32_t temp2;
	uint32_t S0;
	uint32_t S1;

	uint32_t h0, h1, h2, h3, h4, h5, h6, h7;

	uint32_t a, b, c, d, e, f, g, h;
	for (int n = id; n < id + npt; n++)
	{
		h0 = midstate[0];
		h1 = midstate[1];
		h2 = midstate[2];
		h3 = midstate[3];
		h4 = midstate[4];
		h5 = midstate[5];
		h6 = midstate[6];
		h7 = midstate[7];

		a = h0;
		b = h1;
		c = h2;
		d = h3;
		e = h4;
		f = h5;
		g = h6;
		h = h7;

		buffer[11] = n;
		// printf("Nonce being used: %d\n" + buffer[11]);

#pragma unroll
		for (j = 0; j < 16; j++)
		{
			block[j] = buffer[j];
		}
#pragma unroll
		for (j = 16; j < 64; j++)
		{
			block[j] = block[j - 16] + block[j - 7] + SIG1c(block[j - 2]) + SIG0c(block[j - 15]);
		}
#pragma unroll
		for (j = 0; j < 64; j++)
		{
			S1 = (ror(e, 6)) ^ (ror(e, 11)) ^ (ror(e, 25));
			temp1 = h + S1 + ((e & f) ^ ((~e) & g)) + k[j] + block[j];
			S0 = (ror(a, 2)) ^ (ror(a, 13)) ^ (ror(a, 22));
			temp2 = S0 + (((a & b) ^ (a & c) ^ (b & c)));

			h = g;
			g = f;
			f = e;
			e = d + temp1;
			d = c;
			c = b;
			b = a;
			a = temp1 + temp2;
		}

		h0 += a;
		h1 += b;
		h2 += c;
		h3 += d;
		h4 += e;
		h5 += f;
		h6 += g;
		h7 += h;

		block[0] = h0;
		block[1] = h1;
		block[2] = h2;
		block[3] = h3;
		block[4] = h4;
		block[5] = h5;
		block[6] = h6;
		block[7] = h7;
		block[8] = 0x80000000;
		block[9] = 0x00000000;
		block[10] = 0x00000000;
		block[11] = 0x00000000;
		block[12] = 0x00000000;
		block[13] = 0x00000000;
		block[14] = 0x00000000;
		block[15] = 0x00000100;

		h0 = a = 0x6a09e667;
		h1 = b = 0xbb67ae85;
		h2 = c = 0x3c6ef372;
		h3 = d = 0xa54ff53a;
		h4 = e = 0x510e527f;
		h5 = f = 0x9b05688c;
		h6 = g = 0x1f83d9ab;
		h7 = h = 0x5be0cd19;

#pragma unroll
		for (j = 16; j < 64; j++)
		{
			block[j] = block[j - 16] + block[j - 7] + SIG1c(block[j - 2]) + SIG0c(block[j - 15]);
		}

#pragma unroll
		for (j = 0; j < 64; j++)
		{
			S1 = (ror(e, 6)) ^ (ror(e, 11)) ^ (ror(e, 25));
			temp1 = h + S1 + ((e & f) ^ ((~e) & g)) + k[j] + block[j];
			S0 = (ror(a, 2)) ^ (ror(a, 13)) ^ (ror(a, 22));
			temp2 = S0 + (((a & b) ^ (a & c) ^ (b & c)));

			h = g;
			g = f;
			f = e;
			e = d + temp1;
			d = c;
			c = b;
			b = a;
			a = temp1 + temp2;
		}

		h0 += a;
		h1 += b;
		h2 += c;
		h3 += d;
		h4 += e;
		h5 += f;
		h6 += g;
		h7 += h;

		if ((h0 == 0) && (h1 <= hextarpowpo))
	//	if ((h0 <= 0x0000000f) && (h1 <= 0xffffffff))
		{
			*nonceOut = n;
			/* Uncomment these for additional mining information/verbosity */
	//		printf("    The hash is: \n%08x %08x %08x %08x %08x %08x %08x %08x \n Target Pow check: %08x \n", h0, h1, h2, h3, h4, h5, h6, h7, hextarpowpo);
		
		}
	}
}

#define headerSize 176

unsigned char* hexToByteArray(const char* hexstring)
{
	size_t len = headerSize * 2;
	size_t final_len = len / 2;
	unsigned char* chrs = (unsigned char*)malloc((final_len) * sizeof(*chrs));
	for (size_t i = 0, j = 0; j<final_len; i += 2, j++)
		chrs[j] = (hexstring[i] % 32 + 9) % 25 * 16 + (hexstring[i + 1] % 32 + 9) % 25;
	return chrs;
}

char hex[headerSize * 2 + 1];
int increment = 0;
int callInc = 0;
char old[4] = { 0x00, 0x00, 0x00, 0x00 };


bool different(char* one, char* two, int length)
{
	int i = 0;
	for (; i < length; i++)
	{
		if (one[i] != two[i]) return true;
	}
	return false;
}

int deviceToUse = 0;
void getHeaderForWork(uint8_t *header)
{
	if (callInc % 200 == 0)
	{
		callInc = 0;
		FILE *fr;

		char fileName[16] = "headeroutXX.txt";
		fileName[9] = (deviceToUse / 10) + 48;
		fileName[10] = (deviceToUse % 10) + 48;

		fr = fopen(fileName, "rt");
		fgets(hex, 352, fr);
		hex[headerSize*2] = '\0';

		if (different(old, hex, 4))
		{
			old[0] = hex[0];
			old[1] = hex[1];
			old[2] = hex[2];
			old[3] = hex[3];

			printf(" Real: %s\n", hex);
		}

		fclose(fr);
	}

	callInc++;
	unsigned char* bufferHeader = hexToByteArray(hex);
	memcpy(header, bufferHeader, headerSize);
	free(bufferHeader);
	increment++;
	long time = std::time(0);

	header[168] = (time & 0x000000FF);
	header[169] = (time & 0x0000FF00) >> 8;
	header[170] = (time & 0x00FF0000) >> 16;
	header[171] = (time & 0xFF000000) >> 24;
}

int hash = 0;
int totalNonces = 0;
int offset = 0;
void grindNonces()
{
	static bool init = false;
	static uint32_t *nonceOut = nullptr;
	static uint8_t *blockHeader = nullptr;
	static uint8_t *midState = nullptr;
	hipError_t ret;

	if (!init)
	{
		ret = hipHostMalloc(&nonceOut, 4);
		if (ret != hipSuccess)
		{
			printf("ERROR ALLOCATION\n");
		}
		ret = hipHostMalloc(&blockHeader, 64);
		if (ret != hipSuccess)
		{
			printf("ERROR ALLOCATION\n");
		}
		ret = hipHostMalloc(&midState, 32);
		if (ret != hipSuccess)
		{
			printf("ERROR ALLOCATION\n");
		}
		init = true;
	}

	int i;

	getHeaderForWork(blockHeader);
	*nonceOut = 0;

	uint32_t block[64];

	uint32_t h0 = 0x6a09e667;
	uint32_t h1 = 0xbb67ae85;
	uint32_t h2 = 0x3c6ef372;
	uint32_t h3 = 0xa54ff53a;
	uint32_t h4 = 0x510e527f;
	uint32_t h5 = 0x9b05688c;
	uint32_t h6 = 0x1f83d9ab;
	uint32_t h7 = 0x5be0cd19;

	uint32_t a = h0;
	uint32_t b = h1;
	uint32_t c = h2;
	uint32_t d = h3;
	uint32_t e = h4;
	uint32_t f = h5;
	uint32_t g = h6;
	uint32_t h = h7;

	for (int i = 0; i < 16; i++)
	{
		block[i] = ((uint32_t)blockHeader[i * 4 + 0] << 24) | ((uint32_t)blockHeader[i * 4 + 1] << 16) | ((uint32_t)blockHeader[i * 4 + 2] << 8) | ((uint32_t)blockHeader[i * 4 + 3]);
	}

	for (int i = 16; i < 64; i++)
	{
		block[i] = block[i - 16] + block[i - 7] + SIG1(block[i - 2]) + SIG0(block[i - 15]);
	}

	uint32_t temp1;
	uint32_t temp2;
	uint32_t S1;
	uint32_t S0;

	for (int i = 0; i < 64; i++)
	{
		S1 = (ROTRIGHT(e, 6)) ^ (ROTRIGHT(e, 11)) ^ (ROTRIGHT(e, 25));
		temp1 = h + S1 + ((e & f) ^ ((~e) & g)) + k[i] + block[i];
		S0 = (ROTRIGHT(a, 2)) ^ (ROTRIGHT(a, 13)) ^ (ROTRIGHT(a, 22));
		temp2 = S0 + (((a & b) ^ (a & c) ^ (b & c)));

		h = g;
		g = f;
		f = e;
		e = d + temp1;
		d = c;
		c = b;
		b = a;
		a = temp1 + temp2;
	}

	h0 += a;
	h1 += b;
	h2 += c;
	h3 += d;
	h4 += e;
	h5 += f;
	h6 += g;
	h7 += h;

	a = h0;
	b = h1;
	c = h2;
	d = h3;
	e = h4;
	f = h5;
	g = h6;
	h = h7;

	for (int i = 0; i < 16; i++)
	{
		block[i] = ((uint32_t)blockHeader[(i + 16) * 4 + 0] << 24) | ((uint32_t)blockHeader[(i + 16) * 4 + 1] << 16) | ((uint32_t)blockHeader[(i + 16) * 4 + 2] << 8) | ((uint32_t)blockHeader[(i + 16) * 4 + 3]);
	}

	for (int i = 16; i < 64; i++)
	{
		block[i] = block[i - 16] + block[i - 7] + SIG1(block[i - 2]) + SIG0(block[i - 15]);
	}

	for (int i = 0; i < 64; i++)
	{
		S1 = (ROTRIGHT(e, 6)) ^ (ROTRIGHT(e, 11)) ^ (ROTRIGHT(e, 25));
		temp1 = h + S1 + ((e & f) ^ ((~e) & g)) + k[i] + block[i];
		S0 = (ROTRIGHT(a, 2)) ^ (ROTRIGHT(a, 13)) ^ (ROTRIGHT(a, 22));
		temp2 = S0 + (((a & b) ^ (a & c) ^ (b & c)));

		h = g;
		g = f;
		f = e;
		e = d + temp1;
		d = c;
		c = b;
		b = a;
		a = temp1 + temp2;
	}

	h0 += a;
	h1 += b;
	h2 += c;
	h3 += d;
	h4 += e;
	h5 += f;
	h6 += g;
	h7 += h;

	uint32_t midstateInternal[8];

	midstateInternal[0] = h0;
	midstateInternal[1] = h1;
	midstateInternal[2] = h2;
	midstateInternal[3] = h3;
	midstateInternal[4] = h4;
	midstateInternal[5] = h5;
	midstateInternal[6] = h6;
	midstateInternal[7] = h7;

	uint32_t remainingHeader[17];
	for (int i = 0; i < 12; i++)
	{
		remainingHeader[i] = ((uint32_t)blockHeader[(i + 32) * 4 + 0] << 24) | ((uint32_t)blockHeader[(i + 32) * 4 + 1] << 16) | ((uint32_t)blockHeader[(i + 32) * 4 + 2] << 8) | ((uint32_t)blockHeader[(i + 32) * 4 + 3]);
	}
	remainingHeader[12] = 0x80000000;
	remainingHeader[13] = 0x00000000;
	remainingHeader[14] = 0x00000000;
	remainingHeader[15] = 0x00000580;


	remainingHeader[16] = ++offset;

	hipMemcpyAsync(midStatemobj, midstateInternal, 32, hipMemcpyHostToDevice, cudastream);
	hipMemcpyAsync(nonceOutmobj, nonceOut, 4, hipMemcpyHostToDevice, cudastream);
	hipMemcpyAsync(blockHeadermobj, remainingHeader, 68, hipMemcpyHostToDevice, cudastream);
	nonceGrindc << < 128, 1024, blocksize, cudastream >> > (blockHeadermobj, midStatemobj, nonceOutmobj, hextarpow);
	hipMemcpyAsync(nonceOut, nonceOutmobj, 4, hipMemcpyDeviceToHost, cudastream);
	hipStreamSynchronize(cudastream);

	if (*nonceOut != 0)
	{
		uint32_t nonce = *nonceOut;
		nonce = (((nonce & 0xFF000000) >> 24) | ((nonce & 0x00FF0000) >> 8) | ((nonce & 0x0000FF00) << 8) | ((nonce & 0x000000FF) << 24));
		uint32_t timestamp = remainingHeader[10];
		timestamp = ((timestamp & 0x000000FF) << 24) + ((timestamp & 0x0000FF00) << 8) + ((timestamp & 0x00FF0000) >> 8) + ((timestamp & 0xFF000000) >> 24);

		FILE* f2;

		char fileName[13] = "datainXX.txt";
		fileName[6] = (deviceToUse / 10) + 48;
		fileName[7] = (deviceToUse % 10) + 48;

		f2 = fopen(fileName, "w");
		while (f2 == NULL)
		{
			f2 = fopen(fileName, "w");
		}

		fprintf(f2, "\$%08x\n", nonce);
		fprintf(f2, "\$%08x", timestamp);
		fclose(f2);


		*nonceOut = 0;
		totalNonces++;
	}

}

static void tpow()
{
	while (true)
	{
		char hex2[11];
		FILE *fr2;

		char fileName[13] = "targetpow.tp";

		fr2 = fopen(fileName, "rt");
		fgets(hex2, 11, fr2);
		for (int i = 0; i < 11; i++)
		{
			tarpow[i] = hex2[i];
		}
		int num = (int)strtol(tarpow, NULL, 10);

		hextarpow = static_cast<uint32_t>(num);
		//printf("%x\n",hextarpow);
		fclose(fr2);
		Sleep(200);
	}
}
int main(int argc, char *argv[])
{
	HANDLE Th1;
	DWORD ThreadID;

	Th1 = CreateThread(NULL, 0, (LPTHREAD_START_ROUTINE)tpow, NULL, 0, &ThreadID);

	int i = 0; int j = 0;

	if (argc > 1)
	{
		for (i = 1; i < argc; i++)
		{
			char* argument = argv[i];
			if (argument[0] == 'd')
			{
				deviceToUse = argument[1] - 48;
			}
		}
	}

	printf("Using Device: %d\n\n", deviceToUse);

	unsigned int items_per_iter = 256 * 256 * 256 * 8;

	unsigned int cycles_per_iter = 15;
	double seconds_per_iter = 10.0;

	int version, ret;
	ret = hipDriverGetVersion(&version);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}

	int deviceCount;
	ret = hipGetDeviceCount(&deviceCount);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}

	hipDeviceProp_t deviceProp;

	printf("CUDA Version: %.1f\n", ((float)version / 1000));
	printf("CUDA Devices: %d\n", deviceCount);

	printf("\n");

	for (int count = 0; count < deviceCount; count++)
	{
		ret = hipGetDeviceProperties(&deviceProp, count);
		if (ret != hipSuccess)
		{
			printf("ERROR ALLOCATION\n");
		}
		printf("Device #%d (%s):\n", count, deviceProp.name);
		printf("    Clock Rate:              %d MHz\n", (deviceProp.clockRate / 1024));
		printf("    Is Integrated:           %s\n", (deviceProp.integrated == 0 ? "false" : "true"));
		printf("    Compute Capability:      %d.%d\n", deviceProp.major, deviceProp.minor);
		printf("    Kernel Concurrency:      %d\n", deviceProp.concurrentKernels);
		printf("    Max Grid Size:           %d\n", deviceProp.maxGridSize);
		printf("    Max Threads per Block:   %d\n", deviceProp.maxThreadsPerBlock);
		printf("    Registers per Block:     %d\n", deviceProp.regsPerBlock);
		printf("    Registers per SM:        %d\n", deviceProp.regsPerMultiprocessor);
		printf("    Processor Count:         %d\n", deviceProp.multiProcessorCount);
		printf("    Shared Memory/Block:     %d\n", deviceProp.sharedMemPerBlock);
		printf("    Shared Memory/Proc:      %d\n", deviceProp.sharedMemPerMultiprocessor);
		printf("    Warp Size:               %d\n", deviceProp.warpSize);
		printf("\n");
	}

	printf("Mining on device #%d...\n\n", deviceToUse);
	ret = hipSetDevice(deviceToUse);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}
	hipDeviceReset();
	ret = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}
	ret = hipStreamCreate(&cudastream);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}
	ret = hipMalloc(&blockHeadermobj, 68);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}
	ret = hipMalloc(&midStatemobj, 32);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}
	ret = hipMalloc(&nonceOutmobj, 4);
	if (ret != hipSuccess)
	{
		printf("ERROR ALLOCATION\n");
	}


	hipError_t e = hipGetLastError();
	printf("Last error: %s\n", hipGetErrorString(e));

	bool quit = false;

	while (!quit)
	{
		grindNonces();
	}
}
